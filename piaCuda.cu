#include "hip/hip_runtime.h"
#include "piaCuda.h"

extern "C"
__device__ int findSpan(int degree, int n, float t, float* knot) {
    if (t == knot[n+1]) {
        return n;
    }
    int low = degree;
    int high = n + 1;
    int mid;

    while (high - low > 1) {
        mid = (high + low) >> 1;
        if (t < knot[mid]-eps)
            high = mid;
        else
            low = mid;
    }
    return low;
}

extern "C"
__device__ void computeBasis(int degree, int span, float param, float* knot, float* basis) {
    float left[4] = {0};
    float right[4] = {0};
    basis[0] = 1.0;
    for (int j = 1; j <= degree; ++j) {
        left[j] = param - knot[span + 1 - j];
        right[j] = knot[span + j] - param;
        float saved = 0.0;
        for (int r = 0; r < j; ++r) {
            float basisMultiplier = basis[r] / (right[r + 1] + left[j - r]);
            basis[r] = saved + right[r + 1] * basisMultiplier;
            saved = left[j - r] * basisMultiplier;
        }
        basis[j] = saved;
    }
}

extern "C" 
void myTranslation(float* ctrlpts, int numberOfPoints, float tx, float ty, float tz) {
    for (int i=0;i<numberOfPoints;i++) {
        ctrlpts[i*3] += tx;
        ctrlpts[i*3+1] += ty;
        ctrlpts[i*3+2] += tz;
    }
    return;
}

extern "C"
void clearMatrix(float* mat, int numberOfPoints) {
    for (int i=0;i<numberOfPoints;i++) {
        mat[i*3] = 0;
        mat[i*3+1] = 0;
        mat[i*3+2] = 0;
    }
    return;
}

// NURBS surface evaluation
extern "C"
__global__ void evaluationGPU(float* knot, float* u, float* v, float* controlPoints, float* result,
int numberOfPointCloud, int numberControlPointUDirection, int numberControlPointVDirection) {
    int id = blockDim.x*blockIdx.x+threadIdx.x;
    if (id>=numberOfPointCloud) {
        return;
    }
    float r1 = 0, r2 = 0, r3 = 0;

    int spanU = findSpan(3, numberControlPointUDirection-1, u[id], knot);
    int spanV = findSpan(3, numberControlPointVDirection-1, v[id], knot);
    
    float basisU[4] = {0};
    float basisV[4] = {0};
    computeBasis(3, spanU, u[id], knot, basisU);
    computeBasis(3, spanV, v[id], knot, basisV);
    int uind = spanU - 3;
    int vind = spanV - 3;
    
    float temp[4][3];
    for (int l=0;l<=3;l++) {
        temp[l][0] = 0.0;
        temp[l][1] = 0.0;
        temp[l][2] = 0.0;
        vind = spanV-3+l;
        for (int k=0;k<=3;k++) {
            temp[l][0] = temp[l][0] + basisU[k]*controlPoints[(vind*numberControlPointUDirection+uind+k)*3];
            temp[l][1] = temp[l][1] + basisU[k]*controlPoints[(vind*numberControlPointUDirection+uind+k)*3+1];
            temp[l][2] = temp[l][2] + basisU[k]*controlPoints[(vind*numberControlPointUDirection+uind+k)*3+2];
        }
    }
    for (int q=0;q<=3;q++) {
        r1 += basisV[q]*temp[q][0];
        r2 += basisV[q]*temp[q][1];
        r3 += basisV[q]*temp[q][2];
    }
    result[id*3] = r1;
    result[id*3+1] = r2;
    result[id*3+2] = r3;
}

extern "C"
void Multiply(float* m, float* t, int pointNumber) {
    for (int i=0;i<pointNumber;i++) {
        float x = t[0]*m[i*3]+t[1]*m[i*3+1]+t[2]*m[i*3+2]+t[3];
        float y = t[4]*m[i*3]+t[5]*m[i*3+1]+t[6]*m[i*3+2]+t[7];
        float z = t[8]*m[i*3]+t[9]*m[i*3+1]+t[10]*m[i*3+2]+t[11];
        m[i*3] = x;
        m[i*3+1] = y;
        m[i*3+2] = z;
    }
}

extern "C"
void Multiply2(float* dest, float* src, float* t, int pointNumber) {
    for (int i=0;i<pointNumber;i++) {
        float x = t[0]*src[i*3]+t[1]*src[i*3+1]+t[2]*src[i*3+2]+t[3];
        float y = t[4]*src[i*3]+t[5]*src[i*3+1]+t[6]*src[i*3+2]+t[7];
        float z = t[8]*src[i*3]+t[9]*src[i*3+1]+t[10]*src[i*3+2]+t[11];
        dest[i*3] = x;
        dest[i*3+1] = y;
        dest[i*3+2] = z;
    }
}

extern "C"
void constrainedPIAGPU(float* knot, int degree, int numberOfPointCloud, int numberControlPointUDirection,
int numberControlPointVDirection, Eigen::MatrixXf& surfacePlusMatrix, Eigen::MatrixXf& surfaceMinusMatrix,
Eigen::VectorXf& U, Eigen::VectorXf& V, Eigen::MatrixXf* controlPointsPlusPIA, Eigen::MatrixXf* controlPointsMinusPIA,
float offsetValue, string modelType) {
    Eigen::MatrixXf evaluationSurfacePlusPIA(numberOfPointCloud, 3);
    Eigen::MatrixXf evaluationSurfaceMinusPIA(numberOfPointCloud, 3);

    // define variables for NURBS surface evaluation
    float* result = (float*)malloc(numberOfPointCloud*3*sizeof(float));
    float* uVector = (float*)malloc(numberOfPointCloud*sizeof(float));
    float* vVector = (float*)malloc(numberOfPointCloud*sizeof(float));

    for (int i=0;i<numberOfPointCloud;i++) {
        uVector[i] = U(i);
        vVector[i] = V(i);
    }

    int iterate = 20;
    int constraintIterationNumber = 6;

    // define the rigid transformation matrices
    float T4g[16], T5g[16], T6g[16], T7g[16];
    float T1d[16], T2d[16], T3d[16], T4d[16];
    float T1p[16], T2p[16], T3p[16], T4p[16];

    T4g[0] = 0;T4g[1] = 0;T4g[2] = 1;T4g[3] = 0.5;
    T4g[4] = 0;T4g[5] = -1;T4g[6] = 0;T4g[7] = 1.5;
    T4g[8] = 1;T4g[9] = 0;T4g[10] = 0;T4g[11] = -0.5;
    T4g[12] = 0;T4g[13] = 0;T4g[14] = 0;T4g[15] = 1;

    T5g[0] = -1;T5g[1] = 0;T5g[2] = 0;T5g[3] = 1.5;
    T5g[4] = 0;T5g[5] = 0;T5g[6] = -1;T5g[7] = 0.5;
    T5g[8] = 0;T5g[9] = -1;T5g[10] = 0;T5g[11] = 0.5;
    T5g[12] = 0;T5g[13] = 0;T5g[14] = 0;T5g[15] = 1;

    T6g[0] = 0;T6g[1] = 0;T6g[2] = -1;T6g[3] = 1.5;
    T6g[4] = 0;T6g[5] = -1;T6g[6] = 0;T6g[7] = 0.5;
    T6g[8] = -1;T6g[9] = 0;T6g[10] = 0;T6g[11] = 1.5;
    T6g[12] = 0;T6g[13] = 0;T6g[14] = 0;T6g[15] = 1;

    T7g[0] = -1;T7g[1] = 0;T7g[2] = 0;T7g[3] = 2.5;
    T7g[4] = 0;T7g[5] = 0;T7g[6] = 1;T7g[7] = 0.5;
    T7g[8] = 0;T7g[9] = 1;T7g[10] = 0;T7g[11] = -0.5;
    T7g[12] = 0;T7g[13] = 0;T7g[14] = 0;T7g[15] = 1;

    T1d[0] = 0;T1d[1] = -1;T1d[2] = 0;T1d[3] = 0;
    T1d[4] = 1;T1d[5] = 0;T1d[6] = 0;T1d[7] = 0;
    T1d[8] = 0;T1d[9] = 0;T1d[10] = -1;T1d[11] = 0;
    T1d[12] = 0;T1d[13] = 0;T1d[14] = 0;T1d[15] = 1;
    
    T2d[0] = -0.5;T2d[1] = -0.5;T2d[2] = sqrt(2)/2.0;T2d[3] = sqrt(2)/2.0;
    T2d[4] = -0.5;T2d[5] = -0.5;T2d[6] = -sqrt(2)/2.0;T2d[7] = sqrt(2)/2.0;
    T2d[8] = sqrt(2)/2.0;T2d[9] = -sqrt(2)/2.0;T2d[10] = 0;T2d[11] = 0;
    T2d[12] = 0;T2d[13] = 0;T2d[14] = 0;T2d[15] = 1;

    T3d[0] = -0.5;T3d[1] = 0.5;T3d[2] = sqrt(2)/2.0;T3d[3] = sqrt(2)/2.0;
    T3d[4] = 0.5;T3d[5] = -0.5;T3d[6] = sqrt(2)/2.0;T3d[7] = -sqrt(2)/2.0;
    T3d[8] = sqrt(2)/2.0;T3d[9] = sqrt(2)/2.0;T3d[10] = 0;T3d[11] = 0;
    T3d[12] = 0;T3d[13] = 0;T3d[14] = 0;T3d[15] = 1;

    T4d[0] = 0;T4d[1] = 1;T4d[2] = 0;T4d[3] = 0;
    T4d[4] = -1;T4d[5] = 0;T4d[6] = 0;T4d[7] = 0;
    T4d[8] = 0;T4d[9] = 0;T4d[10] = -1;T4d[11] = 0;
    T4d[12] = 0;T4d[13] = 0;T4d[14] = 0;T4d[15] = 1;

    T1p[0] = 0.5;T1p[1] = -0.5;T1p[2] = -sqrt(2)/2.0;T1p[3] = -sqrt(2)/4.0;
    T1p[4] = -0.5;T1p[5] = 0.5;T1p[6] = -sqrt(2)/2.0;T1p[7] = -sqrt(2)/4.0;
    T1p[8] = -sqrt(2)/2.0;T1p[9] = -sqrt(2)/2.0;T1p[10] = 0;T1p[11] = -0.5;
    T1p[12] = 0;T1p[13] = 0;T1p[14] = 0;T1p[15] = 1;

    T2p[0] = 0.5;T2p[1] = 0.5;T2p[2] = -sqrt(2)/2.0;T2p[3] = sqrt(2)/4.0;
    T2p[4] = 0.5;T2p[5] = 0.5;T2p[6] = sqrt(2)/2.0;T2p[7] = -sqrt(2)/4.0;
    T2p[8] = -sqrt(2)/2.0;T2p[9] = sqrt(2)/2.0;T2p[10] = 0;T2p[11] = 0.5;
    T2p[12] = 0;T2p[13] = 0;T2p[14] = 0;T2p[15] = 1;

    T3p[0] = 0;T3p[1] = -1;T3p[2] = 0;T3p[3] = 0;
    T3p[4] = -1;T3p[5] = 0;T3p[6] = 0;T3p[7] = 0;
    T3p[8] = 0;T3p[9] = 0;T3p[10] = 1;T3p[11] = 0;
    T3p[12] = 0;T3p[13] = 0;T3p[14] = 0;T3p[15] = 1;

    T4p[0] = 0;T4p[1] = 1;T4p[2] = 0;T4p[3] = 0;
    T4p[4] = 1;T4p[5] = 0;T4p[6] = 0;T4p[7] = 0;
    T4p[8] = 0;T4p[9] = 0;T4p[10] = 1;T4p[11] = 0;
    T4p[12] = 0;T4p[13] = 0;T4p[14] = 0;T4p[15] = 1;
    
    float* firstOrderEdge1 = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_origin = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_second = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_second_origin = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_second_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge1_second_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge2 = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_origin = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_second = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_second_origin = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_second_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge2_second_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge3 = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_origin = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_second = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_second_origin = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_second_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge3_second_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge4 = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_origin = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_second = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_second_origin = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_second_minus = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
    float* firstOrderEdge4_second_origin_minus = (float*)malloc((numberControlPointVDirection-4)*3*sizeof(float));
    float* secondOrderEdge1 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_2 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_3 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_second_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_2_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_3_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge1_second_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_2 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_3 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_second_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_2_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_3_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge2_second_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_2 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_3 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_second_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_2_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_3_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge3_second_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_2 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_3 = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_second_origin = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_2_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_3_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
    float* secondOrderEdge4_second_origin_minus = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));

    // control points for two primitive surfaces with opposite offset
    float* controlPointsPlusPIAGPU = (float*)malloc(numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float));
    float* controlPointsMinusPIAGPU = (float*)malloc(numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float));
    
    // constraint
    float* constrainMatrix = (float*)malloc(numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float));
    
    // initialize -- compute P0
    initialzeCtrlPts(numberControlPointUDirection*numberControlPointVDirection, controlPointsPlusPIAGPU, surfacePlusMatrix);
    initialzeCtrlPts(numberControlPointUDirection*numberControlPointVDirection, controlPointsMinusPIAGPU, surfaceMinusMatrix);

    dim3 threadsPerBlock(32);
    dim3 blocksPerGrid(std::ceil(numberOfPointCloud*1.0/threadsPerBlock.x));

    float* d_knot;
    float* d_u;
    float* d_v;
    float* d_controlPoints;
    float* d_result;

    hipError_t cudastatus;
    cudastatus = hipMalloc((void**)&d_knot, (numberControlPointUDirection+degree+1)*sizeof(float));
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > knot allocate error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMalloc((void**)&d_u, numberOfPointCloud*sizeof(float));
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > u allocate error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMalloc((void**)&d_v, numberOfPointCloud*sizeof(float));
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > v allocate error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMalloc((void**)&d_controlPoints, numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float));
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > control points allocate error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMalloc((void**)&d_result, numberOfPointCloud*3*sizeof(float));
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > result allocate error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMemcpy(d_knot, knot, (numberControlPointUDirection+degree+1)*sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > knot transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMemcpy(d_u, uVector, numberOfPointCloud*sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > u transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMemcpy(d_v, vVector, numberOfPointCloud*sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > v transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    cudastatus = hipMemcpy(d_controlPoints, controlPointsPlusPIAGPU, numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > control points 1 transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    // compute C
    evaluationGPU<<<blocksPerGrid, threadsPerBlock>>>(d_knot, d_u, d_v, d_controlPoints, d_result, numberOfPointCloud, numberControlPointUDirection, numberControlPointVDirection);

    cudastatus = hipMemcpy(result, d_result, numberOfPointCloud*3*sizeof(float), hipMemcpyDeviceToHost);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > result 1 transfer D2H error: " << hipGetErrorString(cudastatus) << std::endl;
    }
    
    for (int i=0;i<numberOfPointCloud;i++) {
        evaluationSurfacePlusPIA(i, 0) = result[i*3];
        evaluationSurfacePlusPIA(i, 1) = result[i*3+1];
        evaluationSurfacePlusPIA(i, 2) = result[i*3+2];
    }

    cudastatus = hipMemcpy(d_controlPoints, controlPointsMinusPIAGPU, numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > control points 2 transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
    }

    evaluationGPU<<<blocksPerGrid, threadsPerBlock>>>(d_knot, d_u, d_v, d_controlPoints, d_result, numberOfPointCloud, numberControlPointUDirection, numberControlPointVDirection);
    
    cudastatus = hipMemcpy(result, d_result, numberOfPointCloud*3*sizeof(float), hipMemcpyDeviceToHost);
    if (hipSuccess != cudastatus) {
        std::cout << "TPMS2STEP > result 2 transfer D2H error: " << hipGetErrorString(cudastatus) << std::endl;
    }

    for (int i=0;i<numberOfPointCloud;i++) {
        evaluationSurfaceMinusPIA(i, 0) = result[i*3];
        evaluationSurfaceMinusPIA(i, 1) = result[i*3+1];
        evaluationSurfaceMinusPIA(i, 2) = result[i*3+2];
    }

    int count = 0;
    while (count < iterate) {
        count++;
        // compute P k+1
        clearMatrix(constrainMatrix, numberControlPointUDirection*numberControlPointVDirection);
        if (modelType == "Gyroid") {
            myTranslation(controlPointsPlusPIAGPU, numberControlPointUDirection*numberControlPointVDirection, 1, 0.5, 0);
            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    if (j == numberControlPointVDirection-1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        firstOrderEdge1[(i-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge1[(i-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge1[(i-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge1_second[(i-2)*3] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge1_second[(i-2)*3+1] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge1_second[(i-2)*3+2] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge1_origin[(i-2)*3] = firstOrderEdge1[(i-2)*3]-firstOrderEdge1_second[(i-2)*3];
                        firstOrderEdge1_origin[(i-2)*3+1] = firstOrderEdge1[(i-2)*3+1]-firstOrderEdge1_second[(i-2)*3+1];
                        firstOrderEdge1_origin[(i-2)*3+2] = firstOrderEdge1[(i-2)*3+2]-firstOrderEdge1_second[(i-2)*3+2];
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            secondOrderEdge1[(i-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge1[(i-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge1[(i-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge1_2[(i-3)*3] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge1_2[(i-3)*3+1] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge1_2[(i-3)*3+2] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge1_3[(i-3)*3] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge1_3[(i-3)*3+1] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge1_3[(i-3)*3+2] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge1_origin[(i-3)*3] = 2*secondOrderEdge1[(i-3)*3]-3*secondOrderEdge1_2[(i-3)*3]+secondOrderEdge1_3[(i-3)*3];
                            secondOrderEdge1_origin[(i-3)*3+1] = 2*secondOrderEdge1[(i-3)*3+1]-3*secondOrderEdge1_2[(i-3)*3+1]+secondOrderEdge1_3[(i-3)*3+1];
                            secondOrderEdge1_origin[(i-3)*3+2] = 2*secondOrderEdge1[(i-3)*3+2]-3*secondOrderEdge1_2[(i-3)*3+2]+secondOrderEdge1_3[(i-3)*3+2];
                        }
                    }
                    if (i == numberControlPointUDirection-1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        firstOrderEdge2[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge2[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_second[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                        firstOrderEdge2_second[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                        firstOrderEdge2_second[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                        firstOrderEdge2_origin[(j-2)*3] = firstOrderEdge2[(j-2)*3]-firstOrderEdge2_second[(j-2)*3];
                        firstOrderEdge2_origin[(j-2)*3+1] = firstOrderEdge2[(j-2)*3+1]-firstOrderEdge2_second[(j-2)*3+1];
                        firstOrderEdge2_origin[(j-2)*3+2] = firstOrderEdge2[(j-2)*3+2]-firstOrderEdge2_second[(j-2)*3+2];
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            secondOrderEdge2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                            secondOrderEdge2_2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                            secondOrderEdge2_2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                            secondOrderEdge2_3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3];
                            secondOrderEdge2_3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+1];
                            secondOrderEdge2_3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+2];
                            secondOrderEdge2_origin[(j-3)*3] = 2*secondOrderEdge2[(j-3)*3]-3*secondOrderEdge2_2[(j-3)*3]+secondOrderEdge2_3[(j-3)*3];
                            secondOrderEdge2_origin[(j-3)*3+1] = 2*secondOrderEdge2[(j-3)*3+1]-3*secondOrderEdge2_2[(j-3)*3+1]+secondOrderEdge2_3[(j-3)*3+1];
                            secondOrderEdge2_origin[(j-3)*3+2] = 2*secondOrderEdge2[(j-3)*3+2]-3*secondOrderEdge2_2[(j-3)*3+2]+secondOrderEdge2_3[(j-3)*3+2];
                        }
                    }
                    if (j == 0 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        firstOrderEdge3[(i-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge3[(i-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge3[(i-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge3_second[(i-2)*3] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge3_second[(i-2)*3+1] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge3_second[(i-2)*3+2] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge3_origin[(i-2)*3] = firstOrderEdge3[(i-2)*3]-firstOrderEdge3_second[(i-2)*3];
                        firstOrderEdge3_origin[(i-2)*3+1] = firstOrderEdge3[(i-2)*3+1]-firstOrderEdge3_second[(i-2)*3+1];
                        firstOrderEdge3_origin[(i-2)*3+2] = firstOrderEdge3[(i-2)*3+2]-firstOrderEdge3_second[(i-2)*3+2];
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            secondOrderEdge3[(i-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge3[(i-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge3[(i-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge3_2[(i-3)*3] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge3_2[(i-3)*3+1] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge3_2[(i-3)*3+2] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge3_3[(i-3)*3] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge3_3[(i-3)*3+1] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge3_3[(i-3)*3+2] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge3_origin[(i-3)*3] = 2*secondOrderEdge3[(i-3)*3]-3*secondOrderEdge3_2[(i-3)*3]+secondOrderEdge3_3[(i-3)*3];
                            secondOrderEdge3_origin[(i-3)*3+1] = 2*secondOrderEdge3[(i-3)*3+1]-3*secondOrderEdge3_2[(i-3)*3+1]+secondOrderEdge3_3[(i-3)*3+1];
                            secondOrderEdge3_origin[(i-3)*3+2] = 2*secondOrderEdge3[(i-3)*3+2]-3*secondOrderEdge3_2[(i-3)*3+2]+secondOrderEdge3_3[(i-3)*3+2];
                        }
                    }
                    if (i == 0 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        firstOrderEdge4[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge4[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];

                        firstOrderEdge4_second[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                        firstOrderEdge4_second[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                        firstOrderEdge4_second[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];

                        firstOrderEdge4_origin[(j-2)*3] = firstOrderEdge4[(j-2)*3]-firstOrderEdge4_second[(j-2)*3];
                        firstOrderEdge4_origin[(j-2)*3+1] = firstOrderEdge4[(j-2)*3+1]-firstOrderEdge4_second[(j-2)*3+1];
                        firstOrderEdge4_origin[(j-2)*3+2] = firstOrderEdge4[(j-2)*3+2]-firstOrderEdge4_second[(j-2)*3+2];

                        if (j != 2 && j != numberControlPointVDirection-3) {
                            secondOrderEdge4[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge4[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                            secondOrderEdge4_2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                            secondOrderEdge4_2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                            secondOrderEdge4_3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3];
                            secondOrderEdge4_3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+1];
                            secondOrderEdge4_3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+2];
                            secondOrderEdge4_origin[(j-3)*3] = 2*secondOrderEdge4[(j-3)*3]-3*secondOrderEdge4_2[(j-3)*3]+secondOrderEdge4_3[(j-3)*3];
                            secondOrderEdge4_origin[(j-3)*3+1] = 2*secondOrderEdge4[(j-3)*3+1]-3*secondOrderEdge4_2[(j-3)*3+1]+secondOrderEdge4_3[(j-3)*3+1];
                            secondOrderEdge4_origin[(j-3)*3+2] = 2*secondOrderEdge4[(j-3)*3+2]-3*secondOrderEdge4_2[(j-3)*3+2]+secondOrderEdge4_3[(j-3)*3+2];
                        }
                    }
                }
            }

            // for (int i=0;i<numberControlPointUDirection-4;i++) {
            //     cout << "1: " << firstOrderEdge1[i*3] << " " << firstOrderEdge1[i*3+1] << " " << firstOrderEdge1[i*3+2] << endl;
            // }

            // rigid transformation
            Multiply(firstOrderEdge1, T4g, numberControlPointUDirection-4);
            Multiply(firstOrderEdge1_second, T4g, numberControlPointUDirection-4);
            Multiply(secondOrderEdge1, T4g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge1_2, T4g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge1_3, T4g, numberControlPointUDirection-6);
            Multiply(firstOrderEdge2, T5g, numberControlPointUDirection-4);
            Multiply(firstOrderEdge2_second, T5g, numberControlPointUDirection-4);
            Multiply(secondOrderEdge2, T5g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge2_2, T5g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge2_3, T5g, numberControlPointUDirection-6);
            Multiply(firstOrderEdge3, T6g, numberControlPointUDirection-4);
            Multiply(firstOrderEdge3_second, T6g, numberControlPointUDirection-4);
            Multiply(secondOrderEdge3, T6g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge3_2, T6g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge3_3, T6g, numberControlPointUDirection-6);
            Multiply(firstOrderEdge4, T7g, numberControlPointUDirection-4);
            Multiply(firstOrderEdge4_second, T7g, numberControlPointUDirection-4);
            Multiply(secondOrderEdge4, T7g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge4_2, T7g, numberControlPointUDirection-6);
            Multiply(secondOrderEdge4_3, T7g, numberControlPointUDirection-6);

            // for (int i=0;i<numberControlPointUDirection-4;i++) {
            //     cout << "2: " << firstOrderEdge1[i*3] << " " << firstOrderEdge1[i*3+1] << " " << firstOrderEdge1[i*3+2] << endl;
            // }

            for (int i=0;i<numberControlPointUDirection-4;i++) {
                firstOrderEdge1_second_origin[i*3] = (-1)*(firstOrderEdge1[i*3]-firstOrderEdge1_second[i*3]);
                firstOrderEdge1_second_origin[i*3+1] = (-1)*(firstOrderEdge1[i*3+1]-firstOrderEdge1_second[i*3+1]);
                firstOrderEdge1_second_origin[i*3+2] = (-1)*(firstOrderEdge1[i*3+2]-firstOrderEdge1_second[i*3+2]);
                firstOrderEdge2_second_origin[i*3] = (-1)*(firstOrderEdge2[i*3]-firstOrderEdge2_second[i*3]);
                firstOrderEdge2_second_origin[i*3+1] = (-1)*(firstOrderEdge2[i*3+1]-firstOrderEdge2_second[i*3+1]);
                firstOrderEdge2_second_origin[i*3+2] = (-1)*(firstOrderEdge2[i*3+2]-firstOrderEdge2_second[i*3+2]);
                firstOrderEdge3_second_origin[i*3] = (-1)*(firstOrderEdge3[i*3]-firstOrderEdge3_second[i*3]);
                firstOrderEdge3_second_origin[i*3+1] = (-1)*(firstOrderEdge3[i*3+1]-firstOrderEdge3_second[i*3+1]);
                firstOrderEdge3_second_origin[i*3+2] = (-1)*(firstOrderEdge3[i*3+2]-firstOrderEdge3_second[i*3+2]);
                firstOrderEdge4_second_origin[i*3] = (-1)*(firstOrderEdge4[i*3]-firstOrderEdge4_second[i*3]);
                firstOrderEdge4_second_origin[i*3+1] = (-1)*(firstOrderEdge4[i*3+1]-firstOrderEdge4_second[i*3+1]);
                firstOrderEdge4_second_origin[i*3+2] = (-1)*(firstOrderEdge4[i*3+2]-firstOrderEdge4_second[i*3+2]);
            }
            for (int i=0;i<numberControlPointUDirection-6;i++) {
                secondOrderEdge1_second_origin[i*3] = (-1)*(2*secondOrderEdge1[i*3]-3*secondOrderEdge1_2[i*3]+secondOrderEdge1_3[i*3]);
                secondOrderEdge1_second_origin[i*3+1] = (-1)*(2*secondOrderEdge1[i*3+1]-3*secondOrderEdge1_2[i*3+1]+secondOrderEdge1_3[i*3+1]);
                secondOrderEdge1_second_origin[i*3+2] = (-1)*(2*secondOrderEdge1[i*3+2]-3*secondOrderEdge1_2[i*3+2]+secondOrderEdge1_3[i*3+2]);
                secondOrderEdge2_second_origin[i*3] = (-1)*(2*secondOrderEdge2[i*3]-3*secondOrderEdge2_2[i*3]+secondOrderEdge2_3[i*3]);
                secondOrderEdge2_second_origin[i*3+1] = (-1)*(2*secondOrderEdge2[i*3+1]-3*secondOrderEdge2_2[i*3+1]+secondOrderEdge2_3[i*3+1]);
                secondOrderEdge2_second_origin[i*3+2] = (-1)*(2*secondOrderEdge2[i*3+2]-3*secondOrderEdge2_2[i*3+2]+secondOrderEdge2_3[i*3+2]);
                secondOrderEdge3_second_origin[i*3] = (-1)*(2*secondOrderEdge3[i*3]-3*secondOrderEdge3_2[i*3]+secondOrderEdge3_3[i*3]);
                secondOrderEdge3_second_origin[i*3+1] = (-1)*(2*secondOrderEdge3[i*3+1]-3*secondOrderEdge3_2[i*3+1]+secondOrderEdge3_3[i*3+1]);
                secondOrderEdge3_second_origin[i*3+2] = (-1)*(2*secondOrderEdge3[i*3+2]-3*secondOrderEdge3_2[i*3+2]+secondOrderEdge3_3[i*3+2]);
                secondOrderEdge4_second_origin[i*3] = (-1)*(2*secondOrderEdge4[i*3]-3*secondOrderEdge4_2[i*3]+secondOrderEdge4_3[i*3]);
                secondOrderEdge4_second_origin[i*3+1] = (-1)*(2*secondOrderEdge4[i*3+1]-3*secondOrderEdge4_2[i*3+1]+secondOrderEdge4_3[i*3+1]);
                secondOrderEdge4_second_origin[i*3+2] = (-1)*(2*secondOrderEdge4[i*3+2]-3*secondOrderEdge4_2[i*3+2]+secondOrderEdge4_3[i*3+2]);
            }

            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    if (j == numberControlPointVDirection-2 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge1_second_origin[(numberControlPointUDirection-5-(i-2))*3]-firstOrderEdge1_origin[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge1_second_origin[(numberControlPointUDirection-5-(i-2))*3+1]-firstOrderEdge1_origin[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge1_second_origin[(numberControlPointUDirection-5-(i-2))*3+2]-firstOrderEdge1_origin[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3] = (secondOrderEdge1_second_origin[(numberControlPointUDirection-7-(i-3))*3]-secondOrderEdge1_origin[(i-3)*3])/12.0;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge1_second_origin[(numberControlPointUDirection-7-(i-3))*3+1]-secondOrderEdge1_origin[(i-3)*3+1])/12.0;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge1_second_origin[(numberControlPointUDirection-7-(i-3))*3+2]-secondOrderEdge1_origin[(i-3)*3+2])/12.0;
                        }
                    }
                    if (i == numberControlPointUDirection-2 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge2_second_origin[(numberControlPointUDirection-5-(j-2))*3]-firstOrderEdge2_origin[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge2_second_origin[(numberControlPointUDirection-5-(j-2))*3+1]-firstOrderEdge2_origin[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge2_second_origin[(numberControlPointUDirection-5-(j-2))*3+2]-firstOrderEdge2_origin[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3] = (secondOrderEdge2_second_origin[(numberControlPointUDirection-7-(j-3))*3]-secondOrderEdge2_origin[(j-3)*3])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+1] = (secondOrderEdge2_second_origin[(numberControlPointUDirection-7-(j-3))*3+1]-secondOrderEdge2_origin[(j-3)*3+1])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+2] = (secondOrderEdge2_second_origin[(numberControlPointUDirection-7-(j-3))*3+2]-secondOrderEdge2_origin[(j-3)*3+2])/12.0;
                        }
                    }
                    if (j == 1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge3_second_origin[(numberControlPointUDirection-5-(i-2))*3]-firstOrderEdge3_origin[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge3_second_origin[(numberControlPointUDirection-5-(i-2))*3+1]-firstOrderEdge3_origin[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge3_second_origin[(numberControlPointUDirection-5-(i-2))*3+2]-firstOrderEdge3_origin[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3] = (secondOrderEdge3_second_origin[(numberControlPointUDirection-7-(i-3))*3]-secondOrderEdge3_origin[(i-3)*3])/12.0;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge3_second_origin[(numberControlPointUDirection-7-(i-3))*3+1]-secondOrderEdge3_origin[(i-3)*3+1])/12.0;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge3_second_origin[(numberControlPointUDirection-7-(i-3))*3+2]-secondOrderEdge3_origin[(i-3)*3+2])/12.0;
                        }
                    }
                    if (i == 1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge4_second_origin[(numberControlPointUDirection-5-(j-2))*3]-firstOrderEdge4_origin[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge4_second_origin[(numberControlPointUDirection-5-(j-2))*3+1]-firstOrderEdge4_origin[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge4_second_origin[(numberControlPointUDirection-5-(j-2))*3+2]-firstOrderEdge4_origin[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3] = (secondOrderEdge4_second_origin[(numberControlPointUDirection-7-(j-3))*3]-secondOrderEdge4_origin[(j-3)*3])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+1] = (secondOrderEdge4_second_origin[(numberControlPointUDirection-7-(j-3))*3+1]-secondOrderEdge4_origin[(j-3)*3+1])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+2] = (secondOrderEdge4_second_origin[(numberControlPointUDirection-7-(j-3))*3+2]-secondOrderEdge4_origin[(j-3)*3+2])/12.0;
                        }
                    }
                }
            }
            myTranslation(controlPointsPlusPIAGPU, numberControlPointUDirection*numberControlPointVDirection, -1, -0.5, 0);
        }
        else if (modelType == "Diamond") {
            clearMatrix(constrainMatrix, numberControlPointUDirection*numberControlPointVDirection);
            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    //edge 2
                    if (j == 0 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        firstOrderEdge2[(i-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge2[(i-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2[(i-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_second[(i-2)*3] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge2_second[(i-2)*3+1] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2_second[(i-2)*3+2] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_origin[(i-2)*3] = firstOrderEdge2[(i-2)*3]-firstOrderEdge2_second[(i-2)*3];
                        firstOrderEdge2_origin[(i-2)*3+1] = firstOrderEdge2[(i-2)*3+1]-firstOrderEdge2_second[(i-2)*3+1];
                        firstOrderEdge2_origin[(i-2)*3+2] = firstOrderEdge2[(i-2)*3+2]-firstOrderEdge2_second[(i-2)*3+2];
                        firstOrderEdge2_minus[(i-2)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge2_minus[(i-2)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2_minus[(i-2)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_second_minus[(i-2)*3] = controlPointsMinusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge2_second_minus[(i-2)*3+1] = controlPointsMinusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2_second_minus[(i-2)*3+2] = controlPointsMinusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_origin_minus[(i-2)*3] = firstOrderEdge2_minus[(i-2)*3]-firstOrderEdge2_second_minus[(i-2)*3];
                        firstOrderEdge2_origin_minus[(i-2)*3+1] = firstOrderEdge2_minus[(i-2)*3+1]-firstOrderEdge2_second_minus[(i-2)*3+1];
                        firstOrderEdge2_origin_minus[(i-2)*3+2] = firstOrderEdge2_minus[(i-2)*3+2]-firstOrderEdge2_second_minus[(i-2)*3+2];
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            secondOrderEdge2[(i-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge2[(i-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2[(i-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_2[(i-3)*3] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_2[(i-3)*3+1] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_2[(i-3)*3+2] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_3[(i-3)*3] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_3[(i-3)*3+1] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_3[(i-3)*3+2] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_origin[(i-3)*3] = 2*secondOrderEdge2[(i-3)*3]-3*secondOrderEdge2_2[(i-3)*3]+secondOrderEdge2_3[(i-3)*3];
                            secondOrderEdge2_origin[(i-3)*3+1] = 2*secondOrderEdge2[(i-3)*3+1]-3*secondOrderEdge2_2[(i-3)*3+1]+secondOrderEdge2_3[(i-3)*3+1];
                            secondOrderEdge2_origin[(i-3)*3+2] = 2*secondOrderEdge2[(i-3)*3+2]-3*secondOrderEdge2_2[(i-3)*3+2]+secondOrderEdge2_3[(i-3)*3+2];
                            secondOrderEdge2_minus[(i-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_minus[(i-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_minus[(i-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_2_minus[(i-3)*3] = controlPointsMinusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_2_minus[(i-3)*3+1] = controlPointsMinusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_2_minus[(i-3)*3+2] = controlPointsMinusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_3_minus[(i-3)*3] = controlPointsMinusPIAGPU[((j+2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_3_minus[(i-3)*3+1] = controlPointsMinusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_3_minus[(i-3)*3+2] = controlPointsMinusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_origin_minus[(i-3)*3] = 2*secondOrderEdge2_minus[(i-3)*3]-3*secondOrderEdge2_2_minus[(i-3)*3]+secondOrderEdge2_3_minus[(i-3)*3];
                            secondOrderEdge2_origin_minus[(i-3)*3+1] = 2*secondOrderEdge2_minus[(i-3)*3+1]-3*secondOrderEdge2_2_minus[(i-3)*3+1]+secondOrderEdge2_3_minus[(i-3)*3+1];
                            secondOrderEdge2_origin_minus[(i-3)*3+2] = 2*secondOrderEdge2_minus[(i-3)*3+2]-3*secondOrderEdge2_2_minus[(i-3)*3+2]+secondOrderEdge2_3_minus[(i-3)*3+2];
                        }
                    }
                    //edge 1
                    if (i == numberControlPointUDirection-1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        firstOrderEdge1[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge1[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge1[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge1_second[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                        firstOrderEdge1_second[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                        firstOrderEdge1_second[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                        firstOrderEdge1_origin[(j-2)*3] = firstOrderEdge1[(j-2)*3]-firstOrderEdge1_second[(j-2)*3];
                        firstOrderEdge1_origin[(j-2)*3+1] = firstOrderEdge1[(j-2)*3+1]-firstOrderEdge1_second[(j-2)*3+1];
                        firstOrderEdge1_origin[(j-2)*3+2] = firstOrderEdge1[(j-2)*3+2]-firstOrderEdge1_second[(j-2)*3+2];
                        firstOrderEdge1_minus[(j-2)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge1_minus[(j-2)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge1_minus[(j-2)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge1_second_minus[(j-2)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                        firstOrderEdge1_second_minus[(j-2)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                        firstOrderEdge1_second_minus[(j-2)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                        firstOrderEdge1_origin_minus[(j-2)*3] = firstOrderEdge1_minus[(j-2)*3]-firstOrderEdge1_second_minus[(j-2)*3];
                        firstOrderEdge1_origin_minus[(j-2)*3+1] = firstOrderEdge1_minus[(j-2)*3+1]-firstOrderEdge1_second_minus[(j-2)*3+1];
                        firstOrderEdge1_origin_minus[(j-2)*3+2] = firstOrderEdge1_minus[(j-2)*3+2]-firstOrderEdge1_second_minus[(j-2)*3+2];
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            secondOrderEdge1[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge1[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge1[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge1_2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                            secondOrderEdge1_2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                            secondOrderEdge1_2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                            secondOrderEdge1_3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3];
                            secondOrderEdge1_3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+1];
                            secondOrderEdge1_3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+2];
                            secondOrderEdge1_origin[(j-3)*3] = 2*secondOrderEdge1[(j-3)*3]-3*secondOrderEdge1_2[(j-3)*3]+secondOrderEdge1_3[(j-3)*3];
                            secondOrderEdge1_origin[(j-3)*3+1] = 2*secondOrderEdge1[(j-3)*3+1]-3*secondOrderEdge1_2[(j-3)*3+1]+secondOrderEdge1_3[(j-3)*3+1];
                            secondOrderEdge1_origin[(j-3)*3+2] = 2*secondOrderEdge1[(j-3)*3+2]-3*secondOrderEdge1_2[(j-3)*3+2]+secondOrderEdge1_3[(j-3)*3+2];
                            secondOrderEdge1_minus[(j-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge1_minus[(j-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge1_minus[(j-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge1_2_minus[(j-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                            secondOrderEdge1_2_minus[(j-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                            secondOrderEdge1_2_minus[(j-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                            secondOrderEdge1_3_minus[(j-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-2))*3];
                            secondOrderEdge1_3_minus[(j-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+1];
                            secondOrderEdge1_3_minus[(j-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+2];
                            secondOrderEdge1_origin_minus[(j-3)*3] = 2*secondOrderEdge1_minus[(j-3)*3]-3*secondOrderEdge1_2_minus[(j-3)*3]+secondOrderEdge1_3_minus[(j-3)*3];
                            secondOrderEdge1_origin_minus[(j-3)*3+1] = 2*secondOrderEdge1_minus[(j-3)*3+1]-3*secondOrderEdge1_2_minus[(j-3)*3+1]+secondOrderEdge1_3_minus[(j-3)*3+1];
                            secondOrderEdge1_origin_minus[(j-3)*3+2] = 2*secondOrderEdge1_minus[(j-3)*3+2]-3*secondOrderEdge1_2_minus[(j-3)*3+2]+secondOrderEdge1_3_minus[(j-3)*3+2];
                        }
                    }
                    // edge 4
                    if (j == numberControlPointVDirection-1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        firstOrderEdge4[(i-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge4[(i-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4[(i-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge4_second[(i-2)*3] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge4_second[(i-2)*3+1] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4_second[(i-2)*3+2] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge4_origin[(i-2)*3] = firstOrderEdge4[(i-2)*3]-firstOrderEdge4_second[(i-2)*3];
                        firstOrderEdge4_origin[(i-2)*3+1] = firstOrderEdge4[(i-2)*3+1]-firstOrderEdge4_second[(i-2)*3+1];
                        firstOrderEdge4_origin[(i-2)*3+2] = firstOrderEdge4[(i-2)*3+2]-firstOrderEdge4_second[(i-2)*3+2];
                        firstOrderEdge4_minus[(i-2)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge4_minus[(i-2)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4_minus[(i-2)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge4_second_minus[(i-2)*3] = controlPointsMinusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge4_second_minus[(i-2)*3+1] = controlPointsMinusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4_second_minus[(i-2)*3+2] = controlPointsMinusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge4_origin_minus[(i-2)*3] = firstOrderEdge4_minus[(i-2)*3]-firstOrderEdge4_second_minus[(i-2)*3];
                        firstOrderEdge4_origin_minus[(i-2)*3+1] = firstOrderEdge4_minus[(i-2)*3+1]-firstOrderEdge4_second_minus[(i-2)*3+1];
                        firstOrderEdge4_origin_minus[(i-2)*3+2] = firstOrderEdge4_minus[(i-2)*3+2]-firstOrderEdge4_second_minus[(i-2)*3+2];
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            secondOrderEdge4[(i-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge4[(i-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4[(i-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_2[(i-3)*3] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_2[(i-3)*3+1] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_2[(i-3)*3+2] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_3[(i-3)*3] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_3[(i-3)*3+1] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_3[(i-3)*3+2] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_origin[(i-3)*3] = 2*secondOrderEdge4[(i-3)*3]-3*secondOrderEdge4_2[(i-3)*3]+secondOrderEdge4_3[(i-3)*3];
                            secondOrderEdge4_origin[(i-3)*3+1] = 2*secondOrderEdge4[(i-3)*3+1]-3*secondOrderEdge4_2[(i-3)*3+1]+secondOrderEdge4_3[(i-3)*3+1];
                            secondOrderEdge4_origin[(i-3)*3+2] = 2*secondOrderEdge4[(i-3)*3+2]-3*secondOrderEdge4_2[(i-3)*3+2]+secondOrderEdge4_3[(i-3)*3+2];
                            secondOrderEdge4_minus[(i-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_minus[(i-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_minus[(i-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_2_minus[(i-3)*3] = controlPointsMinusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_2_minus[(i-3)*3+1] = controlPointsMinusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_2_minus[(i-3)*3+2] = controlPointsMinusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_3_minus[(i-3)*3] = controlPointsMinusPIAGPU[((j-2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_3_minus[(i-3)*3+1] = controlPointsMinusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_3_minus[(i-3)*3+2] = controlPointsMinusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_origin_minus[(i-3)*3] = 2*secondOrderEdge4_minus[(i-3)*3]-3*secondOrderEdge4_2_minus[(i-3)*3]+secondOrderEdge4_3_minus[(i-3)*3];
                            secondOrderEdge4_origin_minus[(i-3)*3+1] = 2*secondOrderEdge4_minus[(i-3)*3+1]-3*secondOrderEdge4_2_minus[(i-3)*3+1]+secondOrderEdge4_3_minus[(i-3)*3+1];
                            secondOrderEdge4_origin_minus[(i-3)*3+2] = 2*secondOrderEdge4_minus[(i-3)*3+2]-3*secondOrderEdge4_2_minus[(i-3)*3+2]+secondOrderEdge4_3_minus[(i-3)*3+2];
                        }
                    }
                    // edge 3
                    if (i == 0 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        firstOrderEdge3[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge3[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge3[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge3_second[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                        firstOrderEdge3_second[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                        firstOrderEdge3_second[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                        firstOrderEdge3_origin[(j-2)*3] = firstOrderEdge3[(j-2)*3]-firstOrderEdge3_second[(j-2)*3];
                        firstOrderEdge3_origin[(j-2)*3+1] = firstOrderEdge3[(j-2)*3+1]-firstOrderEdge3_second[(j-2)*3+1];
                        firstOrderEdge3_origin[(j-2)*3+2] = firstOrderEdge3[(j-2)*3+2]-firstOrderEdge3_second[(j-2)*3+2];
                        firstOrderEdge3_minus[(j-2)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge3_minus[(j-2)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge3_minus[(j-2)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge3_second_minus[(j-2)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                        firstOrderEdge3_second_minus[(j-2)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                        firstOrderEdge3_second_minus[(j-2)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                        firstOrderEdge3_origin_minus[(j-2)*3] = firstOrderEdge3_minus[(j-2)*3]-firstOrderEdge3_second_minus[(j-2)*3];
                        firstOrderEdge3_origin_minus[(j-2)*3+1] = firstOrderEdge3_minus[(j-2)*3+1]-firstOrderEdge3_second_minus[(j-2)*3+1];
                        firstOrderEdge3_origin_minus[(j-2)*3+2] = firstOrderEdge3_minus[(j-2)*3+2]-firstOrderEdge3_second_minus[(j-2)*3+2];
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            secondOrderEdge3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge3_2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                            secondOrderEdge3_2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                            secondOrderEdge3_2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                            secondOrderEdge3_3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3];
                            secondOrderEdge3_3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+1];
                            secondOrderEdge3_3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+2];
                            secondOrderEdge3_origin[(j-3)*3] = 2*secondOrderEdge3[(j-3)*3]-3*secondOrderEdge3_2[(j-3)*3]+secondOrderEdge3_3[(j-3)*3];
                            secondOrderEdge3_origin[(j-3)*3+1] = 2*secondOrderEdge3[(j-3)*3+1]-3*secondOrderEdge3_2[(j-3)*3+1]+secondOrderEdge3_3[(j-3)*3+1];
                            secondOrderEdge3_origin[(j-3)*3+2] = 2*secondOrderEdge3[(j-3)*3+2]-3*secondOrderEdge3_2[(j-3)*3+2]+secondOrderEdge3_3[(j-3)*3+2];
                            secondOrderEdge3_minus[(j-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge3_minus[(j-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge3_minus[(j-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge3_2_minus[(j-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                            secondOrderEdge3_2_minus[(j-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                            secondOrderEdge3_2_minus[(j-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                            secondOrderEdge3_3_minus[(j-3)*3] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+2))*3];
                            secondOrderEdge3_3_minus[(j-3)*3+1] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+1];
                            secondOrderEdge3_3_minus[(j-3)*3+2] = controlPointsMinusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+2];
                            secondOrderEdge3_origin_minus[(j-3)*3] = 2*secondOrderEdge3_minus[(j-3)*3]-3*secondOrderEdge3_2_minus[(j-3)*3]+secondOrderEdge3_3_minus[(j-3)*3];
                            secondOrderEdge3_origin_minus[(j-3)*3+1] = 2*secondOrderEdge3_minus[(j-3)*3+1]-3*secondOrderEdge3_2_minus[(j-3)*3+1]+secondOrderEdge3_3_minus[(j-3)*3+1];
                            secondOrderEdge3_origin_minus[(j-3)*3+2] = 2*secondOrderEdge3_minus[(j-3)*3+2]-3*secondOrderEdge3_2_minus[(j-3)*3+2]+secondOrderEdge3_3_minus[(j-3)*3+2];
                        }
                    }
                }
            }
            // rigid transformation
            float* firstOrderEdge4_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge4_second_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge4_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge4_second_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge2_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge2_second_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge2_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge2_second_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge1_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge1_second_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge1_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge1_second_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge3_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge3_second_minus_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge3_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* firstOrderEdge3_second_temp = (float*)malloc((numberControlPointUDirection-4)*3*sizeof(float));
            float* secondOrderEdge1_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge1_2_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge1_3_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge1_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge1_2_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge1_3_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge4_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge4_2_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge4_3_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge4_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge4_2_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge4_3_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge2_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge2_2_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge2_3_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge2_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge2_2_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge2_3_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge3_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge3_2_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge3_3_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge3_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge3_2_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            float* secondOrderEdge3_3_minus_temp = (float*)malloc((numberControlPointUDirection-6)*3*sizeof(float));
            Multiply2(firstOrderEdge4_temp, firstOrderEdge4, T1d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge4_second_temp, firstOrderEdge4_second, T1d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge4_minus_temp, firstOrderEdge4_minus, T1d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge4_second_minus_temp, firstOrderEdge4_second_minus, T1d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge2_temp, firstOrderEdge2, T2d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge2_second_temp, firstOrderEdge2_second, T2d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge2_minus_temp, firstOrderEdge2_minus, T2d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge2_second_minus_temp, firstOrderEdge2_second_minus, T2d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge3_temp, firstOrderEdge3, T3d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge3_second_temp, firstOrderEdge3_second, T3d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge3_minus_temp, firstOrderEdge3_minus, T3d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge3_second_minus_temp, firstOrderEdge3_second_minus, T3d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge1_temp, firstOrderEdge1, T4d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge1_second_temp, firstOrderEdge1_second, T4d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge1_minus_temp, firstOrderEdge1_minus, T4d, numberControlPointUDirection-4);
            Multiply2(firstOrderEdge1_second_minus_temp, firstOrderEdge1_second_minus, T4d, numberControlPointUDirection-4);
            Multiply2(secondOrderEdge1_temp, secondOrderEdge1, T4d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge1_2_temp, secondOrderEdge1_2, T4d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge1_3_temp, secondOrderEdge1_3, T4d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge1_minus_temp, secondOrderEdge1_minus, T4d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge1_2_minus_temp, secondOrderEdge1_2_minus, T4d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge1_3_minus_temp, secondOrderEdge1_3_minus, T4d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge4_temp, secondOrderEdge4, T1d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge4_2_temp, secondOrderEdge4_2, T1d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge4_3_temp, secondOrderEdge4_3, T1d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge4_minus_temp, secondOrderEdge4_minus, T1d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge4_2_minus_temp, secondOrderEdge4_2_minus, T1d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge4_3_minus_temp, secondOrderEdge4_3_minus, T1d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge2_temp, secondOrderEdge2, T2d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge2_2_temp, secondOrderEdge2_2, T2d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge2_3_temp, secondOrderEdge2_3, T2d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge2_minus_temp, secondOrderEdge2_minus, T2d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge2_2_minus_temp, secondOrderEdge2_2_minus, T2d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge2_3_minus_temp, secondOrderEdge2_3_minus, T2d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge3_temp, secondOrderEdge3, T3d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge3_2_temp, secondOrderEdge3_2, T3d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge3_3_temp, secondOrderEdge3_3, T3d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge3_minus_temp, secondOrderEdge3_minus, T3d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge3_2_minus_temp, secondOrderEdge3_2_minus, T3d, numberControlPointUDirection-6);
            Multiply2(secondOrderEdge3_3_minus_temp, secondOrderEdge3_3_minus, T3d, numberControlPointUDirection-6);

            // calculate the vectors after transformation
            for (int i=0;i<numberControlPointUDirection-4;i++) {
                firstOrderEdge1_second_origin[i*3] = (-1)*(firstOrderEdge1_temp[i*3]-firstOrderEdge1_second_temp[i*3]);
                firstOrderEdge1_second_origin[i*3+1] = (-1)*(firstOrderEdge1_temp[i*3+1]-firstOrderEdge1_second_temp[i*3+1]);
                firstOrderEdge1_second_origin[i*3+2] = (-1)*(firstOrderEdge1_temp[i*3+2]-firstOrderEdge1_second_temp[i*3+2]);
                firstOrderEdge1_second_origin_minus[i*3] = (-1)*(firstOrderEdge1_minus_temp[i*3]-firstOrderEdge1_second_minus_temp[i*3]);
                firstOrderEdge1_second_origin_minus[i*3+1] = (-1)*(firstOrderEdge1_minus_temp[i*3+1]-firstOrderEdge1_second_minus_temp[i*3+1]);
                firstOrderEdge1_second_origin_minus[i*3+2] = (-1)*(firstOrderEdge1_minus_temp[i*3+2]-firstOrderEdge1_second_minus_temp[i*3+2]);
                firstOrderEdge2_second_origin[i*3] = (-1)*(firstOrderEdge2_temp[i*3]-firstOrderEdge2_second_temp[i*3]);
                firstOrderEdge2_second_origin[i*3+1] = (-1)*(firstOrderEdge2_temp[i*3+1]-firstOrderEdge2_second_temp[i*3+1]);
                firstOrderEdge2_second_origin[i*3+2] = (-1)*(firstOrderEdge2_temp[i*3+2]-firstOrderEdge2_second_temp[i*3+2]);
                firstOrderEdge2_second_origin_minus[i*3] = (-1)*(firstOrderEdge2_minus_temp[i*3]-firstOrderEdge2_second_minus_temp[i*3]);
                firstOrderEdge2_second_origin_minus[i*3+1] = (-1)*(firstOrderEdge2_minus_temp[i*3+1]-firstOrderEdge2_second_minus_temp[i*3+1]);
                firstOrderEdge2_second_origin_minus[i*3+2] = (-1)*(firstOrderEdge2_minus_temp[i*3+2]-firstOrderEdge2_second_minus_temp[i*3+2]);
                firstOrderEdge3_second_origin[i*3] = (-1)*(firstOrderEdge3_temp[i*3]-firstOrderEdge3_second_temp[i*3]);
                firstOrderEdge3_second_origin[i*3+1] = (-1)*(firstOrderEdge3_temp[i*3+1]-firstOrderEdge3_second_temp[i*3+1]);
                firstOrderEdge3_second_origin[i*3+2] = (-1)*(firstOrderEdge3_temp[i*3+2]-firstOrderEdge3_second_temp[i*3+2]);
                firstOrderEdge3_second_origin_minus[i*3] = (-1)*(firstOrderEdge3_minus_temp[i*3]-firstOrderEdge3_second_minus_temp[i*3]);
                firstOrderEdge3_second_origin_minus[i*3+1] = (-1)*(firstOrderEdge3_minus_temp[i*3+1]-firstOrderEdge3_second_minus_temp[i*3+1]);
                firstOrderEdge3_second_origin_minus[i*3+2] = (-1)*(firstOrderEdge3_minus_temp[i*3+2]-firstOrderEdge3_second_minus_temp[i*3+2]);
                firstOrderEdge4_second_origin[i*3] = (-1)*(firstOrderEdge4_temp[i*3]-firstOrderEdge4_second_temp[i*3]);
                firstOrderEdge4_second_origin[i*3+1] = (-1)*(firstOrderEdge4_temp[i*3+1]-firstOrderEdge4_second_temp[i*3+1]);
                firstOrderEdge4_second_origin[i*3+2] = (-1)*(firstOrderEdge4_temp[i*3+2]-firstOrderEdge4_second_temp[i*3+2]);
                firstOrderEdge4_second_origin_minus[i*3] = (-1)*(firstOrderEdge4_minus_temp[i*3]-firstOrderEdge4_second_minus_temp[i*3]);
                firstOrderEdge4_second_origin_minus[i*3+1] = (-1)*(firstOrderEdge4_minus_temp[i*3+1]-firstOrderEdge4_second_minus_temp[i*3+1]);
                firstOrderEdge4_second_origin_minus[i*3+2] = (-1)*(firstOrderEdge4_minus_temp[i*3+2]-firstOrderEdge4_second_minus_temp[i*3+2]);
            }
            for (int i=0;i<numberControlPointUDirection-6;i++) {
                secondOrderEdge1_second_origin[i*3] = (-1)*(2*secondOrderEdge1_temp[i*3]-3*secondOrderEdge1_2_temp[i*3]+secondOrderEdge1_3_temp[i*3]);
                secondOrderEdge1_second_origin[i*3+1] = (-1)*(2*secondOrderEdge1_temp[i*3+1]-3*secondOrderEdge1_2_temp[i*3+1]+secondOrderEdge1_3_temp[i*3+1]);
                secondOrderEdge1_second_origin[i*3+2] = (-1)*(2*secondOrderEdge1_temp[i*3+2]-3*secondOrderEdge1_2_temp[i*3+2]+secondOrderEdge1_3_temp[i*3+2]);
                secondOrderEdge2_second_origin[i*3] = (-1)*(2*secondOrderEdge2_temp[i*3]-3*secondOrderEdge2_2_temp[i*3]+secondOrderEdge2_3_temp[i*3]);
                secondOrderEdge2_second_origin[i*3+1] = (-1)*(2*secondOrderEdge2_temp[i*3+1]-3*secondOrderEdge2_2_temp[i*3+1]+secondOrderEdge2_3_temp[i*3+1]);
                secondOrderEdge2_second_origin[i*3+2] = (-1)*(2*secondOrderEdge2_temp[i*3+2]-3*secondOrderEdge2_2_temp[i*3+2]+secondOrderEdge2_3_temp[i*3+2]);
                secondOrderEdge3_second_origin[i*3] = (-1)*(2*secondOrderEdge3_temp[i*3]-3*secondOrderEdge3_2_temp[i*3]+secondOrderEdge3_3_temp[i*3]);
                secondOrderEdge3_second_origin[i*3+1] = (-1)*(2*secondOrderEdge3_temp[i*3+1]-3*secondOrderEdge3_2_temp[i*3+1]+secondOrderEdge3_3_temp[i*3+1]);
                secondOrderEdge3_second_origin[i*3+2] = (-1)*(2*secondOrderEdge3_temp[i*3+2]-3*secondOrderEdge3_2_temp[i*3+2]+secondOrderEdge3_3_temp[i*3+2]);
                secondOrderEdge4_second_origin[i*3] = (-1)*(2*secondOrderEdge4_temp[i*3]-3*secondOrderEdge4_2_temp[i*3]+secondOrderEdge4_3_temp[i*3]);
                secondOrderEdge4_second_origin[i*3+1] = (-1)*(2*secondOrderEdge4_temp[i*3+1]-3*secondOrderEdge4_2_temp[i*3+1]+secondOrderEdge4_3_temp[i*3+1]);
                secondOrderEdge4_second_origin[i*3+2] = (-1)*(2*secondOrderEdge4_temp[i*3+2]-3*secondOrderEdge4_2_temp[i*3+2]+secondOrderEdge4_3_temp[i*3+2]);
                secondOrderEdge1_second_origin_minus[i*3] = (-1)*(2*secondOrderEdge1_minus_temp[i*3]-3*secondOrderEdge1_2_minus_temp[i*3]+secondOrderEdge1_3_minus_temp[i*3]);
                secondOrderEdge1_second_origin_minus[i*3+1] = (-1)*(2*secondOrderEdge1_minus_temp[i*3+1]-3*secondOrderEdge1_2_minus_temp[i*3+1]+secondOrderEdge1_3_minus_temp[i*3+1]);
                secondOrderEdge1_second_origin_minus[i*3+2] = (-1)*(2*secondOrderEdge1_minus_temp[i*3+2]-3*secondOrderEdge1_2_minus_temp[i*3+2]+secondOrderEdge1_3_minus_temp[i*3+2]);
                secondOrderEdge2_second_origin_minus[i*3] = (-1)*(2*secondOrderEdge2_minus_temp[i*3]-3*secondOrderEdge2_2_minus_temp[i*3]+secondOrderEdge2_3_minus_temp[i*3]);
                secondOrderEdge2_second_origin_minus[i*3+1] = (-1)*(2*secondOrderEdge2_minus_temp[i*3+1]-3*secondOrderEdge2_2_minus_temp[i*3+1]+secondOrderEdge2_3_minus_temp[i*3+1]);
                secondOrderEdge2_second_origin_minus[i*3+2] = (-1)*(2*secondOrderEdge2_minus_temp[i*3+2]-3*secondOrderEdge2_2_minus_temp[i*3+2]+secondOrderEdge2_3_minus_temp[i*3+2]);
                secondOrderEdge3_second_origin_minus[i*3] = (-1)*(2*secondOrderEdge3_minus_temp[i*3]-3*secondOrderEdge3_2_minus_temp[i*3]+secondOrderEdge3_3_minus_temp[i*3]);
                secondOrderEdge3_second_origin_minus[i*3+1] = (-1)*(2*secondOrderEdge3_minus_temp[i*3+1]-3*secondOrderEdge3_2_minus_temp[i*3+1]+secondOrderEdge3_3_minus_temp[i*3+1]);
                secondOrderEdge3_second_origin_minus[i*3+2] = (-1)*(2*secondOrderEdge3_minus_temp[i*3+2]-3*secondOrderEdge3_2_minus_temp[i*3+2]+secondOrderEdge3_3_minus_temp[i*3+2]);
                secondOrderEdge4_second_origin_minus[i*3] = (-1)*(2*secondOrderEdge4_minus_temp[i*3]-3*secondOrderEdge4_2_minus_temp[i*3]+secondOrderEdge4_3_minus_temp[i*3]);
                secondOrderEdge4_second_origin_minus[i*3+1] = (-1)*(2*secondOrderEdge4_minus_temp[i*3+1]-3*secondOrderEdge4_2_minus_temp[i*3+1]+secondOrderEdge4_3_minus_temp[i*3+1]);
                secondOrderEdge4_second_origin_minus[i*3+2] = (-1)*(2*secondOrderEdge4_minus_temp[i*3+2]-3*secondOrderEdge4_2_minus_temp[i*3+2]+secondOrderEdge4_3_minus_temp[i*3+2]);
            }

            // calculate the constraints for the plus surface
            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    // edge 1
                    if (i == numberControlPointUDirection-2 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge4_second_origin_minus[(j-2)*3]-firstOrderEdge1_origin[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge4_second_origin_minus[(j-2)*3+1]-firstOrderEdge1_origin[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge4_second_origin_minus[(j-2)*3+2]-firstOrderEdge1_origin[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3] = (secondOrderEdge4_second_origin_minus[(j-3)*3]-secondOrderEdge1_origin[(j-3)*3])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+1] = (secondOrderEdge4_second_origin_minus[(j-3)*3+1]-secondOrderEdge1_origin[(j-3)*3+1])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+2] = (secondOrderEdge4_second_origin_minus[(j-3)*3+2]-secondOrderEdge1_origin[(j-3)*3+2])/12.0;
                        }
                    }
                    // edge 2
                    if (j == 1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge2_second_origin_minus[(i-2)*3]-firstOrderEdge2_origin[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge2_second_origin_minus[(i-2)*3+1]-firstOrderEdge2_origin[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge2_second_origin_minus[(i-2)*3+2]-firstOrderEdge2_origin[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3] = (secondOrderEdge2_second_origin_minus[(i-3)*3]-secondOrderEdge2_origin[(i-3)*3])/12.0;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge2_second_origin_minus[(i-3)*3+1]-secondOrderEdge2_origin[(i-3)*3+1])/12.0;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge2_second_origin_minus[(i-3)*3+2]-secondOrderEdge2_origin[(i-3)*3+2])/12.0;
                        }
                    }
                    // edge 3
                    if (i == 1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge3_second_origin_minus[(j-2)*3]-firstOrderEdge3_origin[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge3_second_origin_minus[(j-2)*3+1]-firstOrderEdge3_origin[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge3_second_origin_minus[(j-2)*3+2]-firstOrderEdge3_origin[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3] = (secondOrderEdge3_second_origin_minus[(j-3)*3]-secondOrderEdge3_origin[(j-3)*3])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+1] = (secondOrderEdge3_second_origin_minus[(j-3)*3+1]-secondOrderEdge3_origin[(j-3)*3+1])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+2] = (secondOrderEdge3_second_origin_minus[(j-3)*3+2]-secondOrderEdge3_origin[(j-3)*3+2])/12.0;
                        }
                    }
                    // edge 4
                    if (j == numberControlPointVDirection-2 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge1_second_origin_minus[(i-2)*3]-firstOrderEdge4_origin[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge1_second_origin_minus[(i-2)*3+1]-firstOrderEdge4_origin[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge1_second_origin_minus[(i-2)*3+2]-firstOrderEdge4_origin[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3] = (secondOrderEdge1_second_origin_minus[(i-3)*3]-secondOrderEdge4_origin[(i-3)*3])/12.0;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge1_second_origin_minus[(i-3)*3+1]-secondOrderEdge4_origin[(i-3)*3+1])/12.0;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge1_second_origin_minus[(i-3)*3+2]-secondOrderEdge4_origin[(i-3)*3+2])/12.0;
                        }
                    }
                }
            }
        }
        else if (modelType == "SchwarzP") {
            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    // edge 1
                    if (i == 0 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        firstOrderEdge1[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge1[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge1[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge1_second[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                        firstOrderEdge1_second[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                        firstOrderEdge1_second[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                        firstOrderEdge1_origin[(j-2)*3] = firstOrderEdge1[(j-2)*3]-firstOrderEdge1_second[(j-2)*3];
                        firstOrderEdge1_origin[(j-2)*3+1] = firstOrderEdge1[(j-2)*3+1]-firstOrderEdge1_second[(j-2)*3+1];
                        firstOrderEdge1_origin[(j-2)*3+2] = firstOrderEdge1[(j-2)*3+2]-firstOrderEdge1_second[(j-2)*3+2];
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            secondOrderEdge1[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge1[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge1[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge1_2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3];
                            secondOrderEdge1_2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+1];
                            secondOrderEdge1_2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+1))*3+2];
                            secondOrderEdge1_3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3];
                            secondOrderEdge1_3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+1];
                            secondOrderEdge1_3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i+2))*3+2];
                            secondOrderEdge1_origin[(j-3)*3] = 2*secondOrderEdge1[(j-3)*3]-3*secondOrderEdge1_2[(j-3)*3]+secondOrderEdge1_3[(j-3)*3];
                            secondOrderEdge1_origin[(j-3)*3+1] = 2*secondOrderEdge1[(j-3)*3+1]-3*secondOrderEdge1_2[(j-3)*3+1]+secondOrderEdge1_3[(j-3)*3+1];
                            secondOrderEdge1_origin[(j-3)*3+2] = 2*secondOrderEdge1[(j-3)*3+2]-3*secondOrderEdge1_2[(j-3)*3+2]+secondOrderEdge1_3[(j-3)*3+2];
                        }
                    }
                    // edge 2
                    if (j == 0 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        firstOrderEdge2[(i-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge2[(i-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2[(i-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_second[(i-2)*3] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge2_second[(i-2)*3+1] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge2_second[(i-2)*3+2] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge2_origin[(i-2)*3] = firstOrderEdge2[(i-2)*3]-firstOrderEdge2_second[(i-2)*3];
                        firstOrderEdge2_origin[(i-2)*3+1] = firstOrderEdge2[(i-2)*3+1]-firstOrderEdge2_second[(i-2)*3+1];
                        firstOrderEdge2_origin[(i-2)*3+2] = firstOrderEdge2[(i-2)*3+2]-firstOrderEdge2_second[(i-2)*3+2];
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            secondOrderEdge2[(i-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge2[(i-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2[(i-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_2[(i-3)*3] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_2[(i-3)*3+1] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_2[(i-3)*3+2] = controlPointsPlusPIAGPU[((j+1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_3[(i-3)*3] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge2_3[(i-3)*3+1] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge2_3[(i-3)*3+2] = controlPointsPlusPIAGPU[((j+2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge2_origin[(i-3)*3] = 2*secondOrderEdge2[(i-3)*3]-3*secondOrderEdge2_2[(i-3)*3]+secondOrderEdge2_3[(i-3)*3];
                            secondOrderEdge2_origin[(i-3)*3+1] = 2*secondOrderEdge2[(i-3)*3+1]-3*secondOrderEdge2_2[(i-3)*3+1]+secondOrderEdge2_3[(i-3)*3+1];
                            secondOrderEdge2_origin[(i-3)*3+2] = 2*secondOrderEdge2[(i-3)*3+2]-3*secondOrderEdge2_2[(i-3)*3+2]+secondOrderEdge2_3[(i-3)*3+2];
                        }
                    }
                    // edge 3
                    if (i == numberControlPointUDirection-1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        firstOrderEdge3[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge3[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge3[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge3_second[(j-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                        firstOrderEdge3_second[(j-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                        firstOrderEdge3_second[(j-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                        firstOrderEdge3_origin[(j-2)*3] = firstOrderEdge3[(j-2)*3]-firstOrderEdge3_second[(j-2)*3];
                        firstOrderEdge3_origin[(j-2)*3+1] = firstOrderEdge3[(j-2)*3+1]-firstOrderEdge3_second[(j-2)*3+1];
                        firstOrderEdge3_origin[(j-2)*3+2] = firstOrderEdge3[(j-2)*3+2]-firstOrderEdge3_second[(j-2)*3+2];
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            secondOrderEdge3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge3_2[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3];
                            secondOrderEdge3_2[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+1];
                            secondOrderEdge3_2[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-1))*3+2];
                            secondOrderEdge3_3[(j-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3];
                            secondOrderEdge3_3[(j-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+1];
                            secondOrderEdge3_3[(j-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+(i-2))*3+2];
                            secondOrderEdge3_origin[(j-3)*3] = 2*secondOrderEdge3[(j-3)*3]-3*secondOrderEdge3_2[(j-3)*3]+secondOrderEdge3_3[(j-3)*3];
                            secondOrderEdge3_origin[(j-3)*3+1] = 2*secondOrderEdge3[(j-3)*3+1]-3*secondOrderEdge3_2[(j-3)*3+1]+secondOrderEdge3_3[(j-3)*3+1];
                            secondOrderEdge3_origin[(j-3)*3+2] = 2*secondOrderEdge3[(j-3)*3+2]-3*secondOrderEdge3_2[(j-3)*3+2]+secondOrderEdge3_3[(j-3)*3+2];
                        }
                    }
                    // edge 4
                    if (j == numberControlPointVDirection-1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        firstOrderEdge4[(i-2)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                        firstOrderEdge4[(i-2)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4[(i-2)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge4_second[(i-2)*3] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                        firstOrderEdge4_second[(i-2)*3+1] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                        firstOrderEdge4_second[(i-2)*3+2] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                        firstOrderEdge4_origin[(i-2)*3] = firstOrderEdge4[(i-2)*3]-firstOrderEdge4_second[(i-2)*3];
                        firstOrderEdge4_origin[(i-2)*3+1] = firstOrderEdge4[(i-2)*3+1]-firstOrderEdge4_second[(i-2)*3+1];
                        firstOrderEdge4_origin[(i-2)*3+2] = firstOrderEdge4[(i-2)*3+2]-firstOrderEdge4_second[(i-2)*3+2];
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            secondOrderEdge4[(i-3)*3] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3];
                            secondOrderEdge4[(i-3)*3+1] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4[(i-3)*3+2] = controlPointsPlusPIAGPU[(j*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_2[(i-3)*3] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_2[(i-3)*3+1] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_2[(i-3)*3+2] = controlPointsPlusPIAGPU[((j-1)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_3[(i-3)*3] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3];
                            secondOrderEdge4_3[(i-3)*3+1] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+1];
                            secondOrderEdge4_3[(i-3)*3+2] = controlPointsPlusPIAGPU[((j-2)*numberControlPointUDirection+i)*3+2];
                            secondOrderEdge4_origin[(i-3)*3] = 2*secondOrderEdge4[(i-3)*3]-3*secondOrderEdge4_2[(i-3)*3]+secondOrderEdge4_3[(i-3)*3];
                            secondOrderEdge4_origin[(i-3)*3+1] = 2*secondOrderEdge4[(i-3)*3+1]-3*secondOrderEdge4_2[(i-3)*3+1]+secondOrderEdge4_3[(i-3)*3+1];
                            secondOrderEdge4_origin[(i-3)*3+2] = 2*secondOrderEdge4[(i-3)*3+2]-3*secondOrderEdge4_2[(i-3)*3+2]+secondOrderEdge4_3[(i-3)*3+2];
                        }
                    }
                }
            }
            // rigid transformation
            Multiply(firstOrderEdge1, T1p, numberControlPointUDirection-4);
            Multiply(firstOrderEdge1_second, T1p, numberControlPointUDirection-4);
            Multiply(secondOrderEdge1, T1p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge1_2, T1p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge1_3, T1p, numberControlPointUDirection-6);
            Multiply(firstOrderEdge2, T2p, numberControlPointUDirection-4);
            Multiply(firstOrderEdge2_second, T2p, numberControlPointUDirection-4);
            Multiply(secondOrderEdge2, T2p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge2_2, T2p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge2_3, T2p, numberControlPointUDirection-6);
            Multiply(firstOrderEdge3, T3p, numberControlPointUDirection-4);
            Multiply(firstOrderEdge3_second, T3p, numberControlPointUDirection-4);
            Multiply(secondOrderEdge3, T3p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge3_2, T3p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge3_3, T3p, numberControlPointUDirection-6);
            Multiply(firstOrderEdge4, T4p, numberControlPointUDirection-4);
            Multiply(firstOrderEdge4_second, T4p, numberControlPointUDirection-4);
            Multiply(secondOrderEdge4, T4p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge4_2, T4p, numberControlPointUDirection-6);
            Multiply(secondOrderEdge4_3, T4p, numberControlPointUDirection-6);

            for (int i=0;i<numberControlPointUDirection-4;i++) {
                firstOrderEdge1_second_origin[i*3] = (-1)*(firstOrderEdge1[i*3]-firstOrderEdge1_second[i*3]);
                firstOrderEdge1_second_origin[i*3+1] = (-1)*(firstOrderEdge1[i*3+1]-firstOrderEdge1_second[i*3+1]);
                firstOrderEdge1_second_origin[i*3+2] = (-1)*(firstOrderEdge1[i*3+2]-firstOrderEdge1_second[i*3+2]);
                firstOrderEdge2_second_origin[i*3] = (-1)*(firstOrderEdge2[i*3]-firstOrderEdge2_second[i*3]);
                firstOrderEdge2_second_origin[i*3+1] = (-1)*(firstOrderEdge2[i*3+1]-firstOrderEdge2_second[i*3+1]);
                firstOrderEdge2_second_origin[i*3+2] = (-1)*(firstOrderEdge2[i*3+2]-firstOrderEdge2_second[i*3+2]);
                firstOrderEdge3_second_origin[i*3] = (-1)*(firstOrderEdge3[i*3]-firstOrderEdge3_second[i*3]);
                firstOrderEdge3_second_origin[i*3+1] = (-1)*(firstOrderEdge3[i*3+1]-firstOrderEdge3_second[i*3+1]);
                firstOrderEdge3_second_origin[i*3+2] = (-1)*(firstOrderEdge3[i*3+2]-firstOrderEdge3_second[i*3+2]);
                firstOrderEdge4_second_origin[i*3] = (-1)*(firstOrderEdge4[i*3]-firstOrderEdge4_second[i*3]);
                firstOrderEdge4_second_origin[i*3+1] = (-1)*(firstOrderEdge4[i*3+1]-firstOrderEdge4_second[i*3+1]);
                firstOrderEdge4_second_origin[i*3+2] = (-1)*(firstOrderEdge4[i*3+2]-firstOrderEdge4_second[i*3+2]);
            }
            for (int i=0;i<numberControlPointUDirection-6;i++) {
                secondOrderEdge1_second_origin[i*3] = (-1)*(2*secondOrderEdge1[i*3]-3*secondOrderEdge1_2[i*3]+secondOrderEdge1_3[i*3]);
                secondOrderEdge1_second_origin[i*3+1] = (-1)*(2*secondOrderEdge1[i*3+1]-3*secondOrderEdge1_2[i*3+1]+secondOrderEdge1_3[i*3+1]);
                secondOrderEdge1_second_origin[i*3+2] = (-1)*(2*secondOrderEdge1[i*3+2]-3*secondOrderEdge1_2[i*3+2]+secondOrderEdge1_3[i*3+2]);
                secondOrderEdge2_second_origin[i*3] = (-1)*(2*secondOrderEdge2[i*3]-3*secondOrderEdge2_2[i*3]+secondOrderEdge2_3[i*3]);
                secondOrderEdge2_second_origin[i*3+1] = (-1)*(2*secondOrderEdge2[i*3+1]-3*secondOrderEdge2_2[i*3+1]+secondOrderEdge2_3[i*3+1]);
                secondOrderEdge2_second_origin[i*3+2] = (-1)*(2*secondOrderEdge2[i*3+2]-3*secondOrderEdge2_2[i*3+2]+secondOrderEdge2_3[i*3+2]);
                secondOrderEdge3_second_origin[i*3] = (-1)*(2*secondOrderEdge3[i*3]-3*secondOrderEdge3_2[i*3]+secondOrderEdge3_3[i*3]);
                secondOrderEdge3_second_origin[i*3+1] = (-1)*(2*secondOrderEdge3[i*3+1]-3*secondOrderEdge3_2[i*3+1]+secondOrderEdge3_3[i*3+1]);
                secondOrderEdge3_second_origin[i*3+2] = (-1)*(2*secondOrderEdge3[i*3+2]-3*secondOrderEdge3_2[i*3+2]+secondOrderEdge3_3[i*3+2]);
                secondOrderEdge4_second_origin[i*3] = (-1)*(2*secondOrderEdge4[i*3]-3*secondOrderEdge4_2[i*3]+secondOrderEdge4_3[i*3]);
                secondOrderEdge4_second_origin[i*3+1] = (-1)*(2*secondOrderEdge4[i*3+1]-3*secondOrderEdge4_2[i*3+1]+secondOrderEdge4_3[i*3+1]);
                secondOrderEdge4_second_origin[i*3+2] = (-1)*(2*secondOrderEdge4[i*3+2]-3*secondOrderEdge4_2[i*3+2]+secondOrderEdge4_3[i*3+2]);
            }

            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    // edge 1
                    if (i == 1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge1_second_origin[(numberControlPointUDirection-5-(j-2))*3]-firstOrderEdge1_origin[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge1_second_origin[(numberControlPointUDirection-5-(j-2))*3+1]-firstOrderEdge1_origin[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge1_second_origin[(numberControlPointUDirection-5-(j-2))*3+2]-firstOrderEdge1_origin[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3] = (secondOrderEdge1_second_origin[(j-3)*3]-secondOrderEdge1_origin[(j-3)*3])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+1] = (secondOrderEdge1_second_origin[(j-3)*3+1]-secondOrderEdge1_origin[(j-3)*3+1])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+2] = (secondOrderEdge1_second_origin[(j-3)*3+2]-secondOrderEdge1_origin[(j-3)*3+2])/12.0;
                        }
                    }
                    // edge 2
                    if (j == 1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge2_second_origin[(i-2)*3]-firstOrderEdge2_origin[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge2_second_origin[(i-2)*3+1]-firstOrderEdge2_origin[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge2_second_origin[(i-2)*3+2]-firstOrderEdge2_origin[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3] = (secondOrderEdge2_second_origin[(i-3)*3]-secondOrderEdge2_origin[(i-3)*3])/12.0;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge2_second_origin[(i-3)*3+1]-secondOrderEdge2_origin[(i-3)*3+1])/12.0;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge2_second_origin[(i-3)*3+2]-secondOrderEdge2_origin[(i-3)*3+2])/12.0;
                        }
                    }
                    // edge 3
                    if (i == numberControlPointUDirection-2 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge3_second_origin[(j-2)*3]-firstOrderEdge3_origin[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge3_second_origin[(j-2)*3+1]-firstOrderEdge3_origin[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge3_second_origin[(j-2)*3+2]-firstOrderEdge3_origin[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3] = (secondOrderEdge3_second_origin[(j-3)*3]-secondOrderEdge3_origin[(j-3)*3])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+1] = (secondOrderEdge3_second_origin[(j-3)*3+1]-secondOrderEdge3_origin[(j-3)*3+1])/12.0;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+2] = (secondOrderEdge3_second_origin[(j-3)*3+2]-secondOrderEdge3_origin[(j-3)*3+2])/12.0;
                        }
                    }
                    // edge 4
                    if (j == numberControlPointVDirection-2 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge4_second_origin[(i-2)*3]-firstOrderEdge4_origin[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge4_second_origin[(i-2)*3+1]-firstOrderEdge4_origin[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge4_second_origin[(i-2)*3+2]-firstOrderEdge4_origin[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3] = (secondOrderEdge4_second_origin[(i-3)*3]-secondOrderEdge4_origin[(i-3)*3])/12.0;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge4_second_origin[(i-3)*3+1]-secondOrderEdge4_origin[(i-3)*3+1])/12.0;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge4_second_origin[(i-3)*3+2]-secondOrderEdge4_origin[(i-3)*3+2])/12.0;
                        }
                    }
                }
            }
        }
        
        if (count>=constraintIterationNumber) {
            for (int i=0;i<numberControlPointUDirection*numberControlPointVDirection;i++) {
                controlPointsPlusPIAGPU[i*3] = controlPointsPlusPIAGPU[i*3] + surfacePlusMatrix(i,0) - evaluationSurfacePlusPIA(i,0) + constrainMatrix[i*3];
                controlPointsPlusPIAGPU[i*3+1] = controlPointsPlusPIAGPU[i*3+1] + surfacePlusMatrix(i,1) - evaluationSurfacePlusPIA(i,1) + constrainMatrix[i*3+1];
                controlPointsPlusPIAGPU[i*3+2] = controlPointsPlusPIAGPU[i*3+2] + surfacePlusMatrix(i,2) - evaluationSurfacePlusPIA(i,2) + constrainMatrix[i*3+2];
            }
        } else {
            for (int i=0;i<numberControlPointUDirection*numberControlPointVDirection;i++) {
                controlPointsPlusPIAGPU[i*3] = controlPointsPlusPIAGPU[i*3] + surfacePlusMatrix(i,0) - evaluationSurfacePlusPIA(i,0);
                controlPointsPlusPIAGPU[i*3+1] = controlPointsPlusPIAGPU[i*3+1] + surfacePlusMatrix(i,1) - evaluationSurfacePlusPIA(i,1);
                controlPointsPlusPIAGPU[i*3+2] = controlPointsPlusPIAGPU[i*3+2] + surfacePlusMatrix(i,2) - evaluationSurfacePlusPIA(i,2);
            }
        }
        if (modelType == "Gyroid") {
            clearMatrix(constrainMatrix, numberControlPointUDirection*numberControlPointVDirection);
        } else if (modelType == "Diamond") {
            clearMatrix(constrainMatrix, numberControlPointUDirection*numberControlPointVDirection);
            // calculate the constraints for minus surface
            for (int i=0;i<numberControlPointUDirection;i++) {
                for (int j=0;j<numberControlPointVDirection;j++) {
                    // edge 1
                    if (i == numberControlPointUDirection-2 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge4_second_origin[(j-2)*3]-firstOrderEdge1_origin_minus[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge4_second_origin[(j-2)*3+1]-firstOrderEdge1_origin_minus[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge4_second_origin[(j-2)*3+2]-firstOrderEdge1_origin_minus[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3] = (secondOrderEdge4_second_origin[(j-3)*3]-secondOrderEdge1_origin_minus[(j-3)*3])/2;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+1] = (secondOrderEdge4_second_origin[(j-3)*3+1]-secondOrderEdge1_origin_minus[(j-3)*3+1])/2;
                            constrainMatrix[(j*numberControlPointUDirection+(i-1))*3+2] = (secondOrderEdge4_second_origin[(j-3)*3+2]-secondOrderEdge1_origin_minus[(j-3)*3+2])/2;
                        }
                    }
                    // edge 2
                    if (j == 1 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge2_second_origin[(i-2)*3]-firstOrderEdge2_origin_minus[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge2_second_origin[(i-2)*3+1]-firstOrderEdge2_origin_minus[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge2_second_origin[(i-2)*3+2]-firstOrderEdge2_origin_minus[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3] = (secondOrderEdge2_second_origin[(i-3)*3]-secondOrderEdge2_origin_minus[(i-3)*3])/2;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge2_second_origin[(i-3)*3+1]-secondOrderEdge2_origin_minus[(i-3)*3+1])/2;
                            constrainMatrix[((j+1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge2_second_origin[(i-3)*3+2]-secondOrderEdge2_origin_minus[(i-3)*3+2])/2;
                        }
                    }
                    // edge 3
                    if (i == 1 && j != 0 && j != 1 && j != numberControlPointVDirection-1 && j != numberControlPointVDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge3_second_origin[(j-2)*3]-firstOrderEdge3_origin_minus[(j-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge3_second_origin[(j-2)*3+1]-firstOrderEdge3_origin_minus[(j-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge3_second_origin[(j-2)*3+2]-firstOrderEdge3_origin_minus[(j-2)*3+2])/2;
                        if (j != 2 && j != numberControlPointVDirection-3) {
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3] = (secondOrderEdge3_second_origin[(j-3)*3]-secondOrderEdge3_origin_minus[(j-3)*3])/2;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+1] = (secondOrderEdge3_second_origin[(j-3)*3+1]-secondOrderEdge3_origin_minus[(j-3)*3+1])/2;
                            constrainMatrix[(j*numberControlPointUDirection+(i+1))*3+2] = (secondOrderEdge3_second_origin[(j-3)*3+2]-secondOrderEdge3_origin_minus[(j-3)*3+2])/2;
                        }
                    }
                    // edge 4
                    if (j == numberControlPointVDirection-2 && i != 0 && i != 1 && i != numberControlPointUDirection-1 && i != numberControlPointUDirection-2) {
                        constrainMatrix[(j*numberControlPointUDirection+i)*3] = (-1)*(firstOrderEdge1_second_origin[(i-2)*3]-firstOrderEdge4_origin_minus[(i-2)*3])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+1] = (-1)*(firstOrderEdge1_second_origin[(i-2)*3+1]-firstOrderEdge4_origin_minus[(i-2)*3+1])/2;
                        constrainMatrix[(j*numberControlPointUDirection+i)*3+2] = (-1)*(firstOrderEdge1_second_origin[(i-2)*3+2]-firstOrderEdge4_origin_minus[(i-2)*3+2])/2;
                        if (i != 2 && i != numberControlPointUDirection-3) {
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3] = (secondOrderEdge1_second_origin[(i-3)*3]-secondOrderEdge4_origin_minus[(i-3)*3])/2;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+1] = (secondOrderEdge1_second_origin[(i-3)*3+1]-secondOrderEdge4_origin_minus[(i-3)*3+1])/2;
                            constrainMatrix[((j-1)*numberControlPointUDirection+i)*3+2] = (secondOrderEdge1_second_origin[(i-3)*3+2]-secondOrderEdge4_origin_minus[(i-3)*3+2])/2;
                        }
                    }
                }
            }
        }
        else if (modelType == "SchwarzP") {
            clearMatrix(constrainMatrix, numberControlPointUDirection*numberControlPointVDirection);
        }

        if (modelType != "SchwarzP") {
            if (count>=constraintIterationNumber) {
                for (int i=0;i<numberControlPointUDirection*numberControlPointVDirection;i++) {
                    controlPointsMinusPIAGPU[i*3] = controlPointsMinusPIAGPU[i*3] + surfaceMinusMatrix(i,0) - evaluationSurfaceMinusPIA(i,0) + constrainMatrix[i*3];
                    controlPointsMinusPIAGPU[i*3+1] = controlPointsMinusPIAGPU[i*3+1] + surfaceMinusMatrix(i,1) - evaluationSurfaceMinusPIA(i,1) + constrainMatrix[i*3+1];
                    controlPointsMinusPIAGPU[i*3+2] = controlPointsMinusPIAGPU[i*3+2] + surfaceMinusMatrix(i,2) - evaluationSurfaceMinusPIA(i,2) + constrainMatrix[i*3+2];
                }
            } else {
                for (int i=0;i<numberControlPointUDirection*numberControlPointVDirection;i++) {
                    controlPointsMinusPIAGPU[i*3] = controlPointsMinusPIAGPU[i*3] + surfaceMinusMatrix(i,0) - evaluationSurfaceMinusPIA(i,0);
                    controlPointsMinusPIAGPU[i*3+1] = controlPointsMinusPIAGPU[i*3+1] + surfaceMinusMatrix(i,1) - evaluationSurfaceMinusPIA(i,1);
                    controlPointsMinusPIAGPU[i*3+2] = controlPointsMinusPIAGPU[i*3+2] + surfaceMinusMatrix(i,2) - evaluationSurfaceMinusPIA(i,2);
                }
            }
        }

        // compute C k+1
        blocksPerGrid = std::ceil(numberOfPointCloud*1.0/threadsPerBlock.x);
        cudastatus = hipMemcpy(d_controlPoints, controlPointsPlusPIAGPU, numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float), hipMemcpyHostToDevice);
        if (hipSuccess != cudastatus) {
            std::cout << "control points 1 transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
        }
        evaluationGPU<<<blocksPerGrid, threadsPerBlock>>>(d_knot, d_u, d_v, d_controlPoints, d_result, numberOfPointCloud, numberControlPointUDirection, numberControlPointVDirection);

        cudastatus = hipMemcpy(result, d_result, numberOfPointCloud*3*sizeof(float), hipMemcpyDeviceToHost);
        if (hipSuccess != cudastatus) {
            std::cout << "result 1 transfer D2H error: " << hipGetErrorString(cudastatus) << std::endl;
        }

        for (int i=0;i<numberOfPointCloud;i++) {
            evaluationSurfacePlusPIA(i, 0) = result[i*3];
            evaluationSurfacePlusPIA(i, 1) = result[i*3+1];
            evaluationSurfacePlusPIA(i, 2) = result[i*3+2];
        }

        hipDeviceSynchronize();
        // if (modelType != "SchwarzP") {
        if (true) {
            cudastatus = hipMemcpy(d_controlPoints, controlPointsMinusPIAGPU, numberControlPointUDirection*numberControlPointVDirection*3*sizeof(float), hipMemcpyHostToDevice);
            if (hipSuccess != cudastatus) {
                std::cout << "control points 2 transfer H2D error: " << hipGetErrorString(cudastatus) << std::endl;
            }

            evaluationGPU<<<blocksPerGrid, threadsPerBlock>>>(d_knot, d_u, d_v, d_controlPoints, d_result, numberOfPointCloud, numberControlPointUDirection, numberControlPointVDirection);
            
            cudastatus = hipMemcpy(result, d_result, numberOfPointCloud*3*sizeof(float), hipMemcpyDeviceToHost);
            if (hipSuccess != cudastatus) {
                std::cout << "result 2 transfer D2H error: " << hipGetErrorString(cudastatus) << std::endl;
            }
            if (modelType != "SchwarzP") {
                for (int i=0;i<numberOfPointCloud;i++) {
                    evaluationSurfaceMinusPIA(i, 0) = result[i*3];
                    evaluationSurfaceMinusPIA(i, 1) = result[i*3+1];
                    evaluationSurfaceMinusPIA(i, 2) = result[i*3+2];
                }
            }
        }
        
    }
    for (int i=0;i<numberControlPointUDirection*numberControlPointVDirection;i++) {
        (*controlPointsPlusPIA)(i,0) = controlPointsPlusPIAGPU[i*3];
        (*controlPointsPlusPIA)(i,1) = controlPointsPlusPIAGPU[i*3+1];
        (*controlPointsPlusPIA)(i,2) = controlPointsPlusPIAGPU[i*3+2];
        if (modelType != "SchwarzP") {
            (*controlPointsMinusPIA)(i,0) = controlPointsMinusPIAGPU[i*3];
            (*controlPointsMinusPIA)(i,1) = controlPointsMinusPIAGPU[i*3+1];
            (*controlPointsMinusPIA)(i,2) = controlPointsMinusPIAGPU[i*3+2];
        }
    }

    free(result);
    free(controlPointsPlusPIAGPU);
    free(controlPointsMinusPIAGPU);
    free(uVector);
    free(vVector);

    free(firstOrderEdge1);
    free(firstOrderEdge1_origin);
    free(firstOrderEdge1_second);
    free(firstOrderEdge1_second_origin);
    free(firstOrderEdge2);
    free(firstOrderEdge2_origin);
    free(firstOrderEdge2_second);
    free(firstOrderEdge2_second_origin);
    free(firstOrderEdge3);
    free(firstOrderEdge3_origin);
    free(firstOrderEdge3_second);
    free(firstOrderEdge3_second_origin);
    free(firstOrderEdge4);
    free(firstOrderEdge4_origin);
    free(firstOrderEdge4_second);
    free(firstOrderEdge4_second_origin);
    free(secondOrderEdge1);
    free(secondOrderEdge1_2);
    free(secondOrderEdge1_3);
    free(secondOrderEdge1_origin);
    free(secondOrderEdge1_second_origin);
    free(secondOrderEdge2);
    free(secondOrderEdge2_2);
    free(secondOrderEdge2_3);
    free(secondOrderEdge2_origin);
    free(secondOrderEdge2_second_origin);
    free(secondOrderEdge3);
    free(secondOrderEdge3_2);
    free(secondOrderEdge3_3);
    free(secondOrderEdge3_origin);
    free(secondOrderEdge3_second_origin);
    free(secondOrderEdge4);
    free(secondOrderEdge4_2);
    free(secondOrderEdge4_3);
    free(secondOrderEdge4_origin);
    free(secondOrderEdge4_second_origin);
    cout << "TPMS2STEP > constrained-PIA over" << endl;
}

extern "C"
void memoryAllocation1(float* a, float* b, float* c, float* d, int number) {
    a = (float*)malloc(number*sizeof(float));
    b = (float*)malloc(number*sizeof(float));
    c = (float*)malloc(number*sizeof(float));
    d = (float*)malloc(number*sizeof(float));
    if (!a || !b || !c || !d) {
        cout << "TPMS2STEP > Memory allocation error." << endl;
    }
}

extern "C"
void memoryAllocation2(float* a, float* b, float* c, float* d, float* e, int number) {
    a = (float*)malloc(number*sizeof(float));
    b = (float*)malloc(number*sizeof(float));
    c = (float*)malloc(number*sizeof(float));
    d = (float*)malloc(number*sizeof(float));
    e = (float*)malloc(number*sizeof(float));
}

extern "C"
void myFree1(float* a, float* b, float* c, float* d) {
    if (a) {free(a);}
    if (b) {free(b);}
    if (c) {free(c);}
    if (d) {free(d);}
}

extern "C"
void myFree2(float* a, float* b, float* c, float* d, float* e) {
    if (a) {free(a);}
    if (b) {free(b);}
    if (c) {free(c);}
    if (d) {free(d);}
    if (e) {free(e);}
}

extern "C"
void initialzeCtrlPts(int number, float* ctrlpts, Eigen::MatrixXf& surface) {
    for (int i=0;i<number;i++) {
        ctrlpts[i*3] = surface(i,0);
        ctrlpts[i*3+1] = surface(i,1);
        ctrlpts[i*3+2] = surface(i,2);
    }
}